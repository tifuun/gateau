#include "hip/hip_runtime.h"
#include "InterfaceCUDA.h"

/*! \file Kernels.cu
    \brief Definitions of CUDA kernels for gateau.

    author: Arend Moerman
*/


// DEFINITIONS OF PHYSICAL AND MATHEMATICAL CONSTANTS
#define KB              1.380649E-23f
#define CL              2.9979246E8f
#define HP              6.62607015E-34f
#define PI              3.14159265

// HANDY STUFF
#define DEG2RAD PI/180

// OBSERVATION-INSTRUMENT PARAMETERS
__constant__ float cdt;                     // Timestep
__constant__ float ct_start;                // Starting time
__constant__ float cf_sample;               // Sampling frequency of readout
__constant__ float csqrt_samp;               // Sampling frequency of readout
__constant__ float cGR_factor;              // Factor for GR noise: 2 * Delta / eta_pb
__constant__ int cnt;                       // Number of time evals
__constant__ int cnf_ch;                    // Number of filter freqs
__constant__ int cnum_stage;

// ATMOSPHERE PARAMETERS
__constant__ float ch_column;               // Column height
__constant__ float cv_wind;                 // Windspeed

// TEXTURE MEMORY
texture<float, hipTextureType1D, hipReadModeElementType> tex_filterbank;
texture<float, hipTextureType1D, hipReadModeElementType> tex_eta_ap;
texture<float, hipTextureType1D, hipReadModeElementType> tex_psd_atm;

// CONSTANTS FOR KERNEL LAUNCHES
#define NTHREADS1D      256

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

/////////////////////////////////
//////// HOST FUNCTIONS /////////
/////////////////////////////////

/**
  Check CUDA API error status of call.
 
  Wrapper for finding errors in CUDA API calls.
 
  @param code The errorcode returned from failed API call.
  @param file The file in which failure occured.
  @param line The line in file in which error occured.
  @param abort Exit code upon error.
 */
__host__ inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**
  Write a CUDA device array to a file, for debugging.
  Templated to work with floats and ints.

  @param array Pointer to device array of type T.
  @param s_array Size of array.
  @param name_txt Name of file to write array to. Name is appended with '.txt' by the function itself.
 */
template <typename T>
__host__ void writeArray(T *array, int s_array, std::string name_txt) {
    
    T *h_array = new T[s_array];
    gpuErrchk( hipMemcpy(h_array, array, s_array * sizeof(T), hipMemcpyDeviceToHost) );
    
    std::ofstream myfile (name_txt + ".txt");
    if (myfile.is_open())
    {
        for(int count = 0; count < s_array; count ++){
            myfile << h_array[count] << "\n" ;
        }

        myfile.close();
    }
    else std::cout << "Unable to open file";
    delete[] h_array;
}

__host__ inline float get_jn_noise(float T, float nu) 
{
    return HP * nu / (expf(HP * nu / (KB * T)) - 1);
}

/////////////////////////////////
/////// DEVICE FUNCTIONS ////////
/////////////////////////////////

__device__ __inline__ void time_wrt_to(int thread_index, 
                                                int thread_index_select=0,
                                                long long int time_offset=0)
{
    if(thread_index == thread_index_select) 
    {
        printf("Thread %d at time %llu w.r.t. offset\n", thread_index, clock64() - time_offset);
    }
}

/**
  Cascade a PSD through a reflector system, and couple to a specific parasitic PSD.

  @param P_nu_in PSD of incoming signal to be cascaded.
  @param eta Efficiency term associated with cascade.
  @param T_parasitic Temperature of parasitic source.
  @param nu Frequency in Hz.

  @returns Cascade output PSD.
 */
__device__ __inline__ float rad_trans(float psd_in, 
                                      float eta, 
                                      float psd_parasitic)
{
    return eta * psd_in + (1 - eta) * psd_parasitic;
}

/**
  Initialize CUDA.
 
  Instantiate program and populate constant memory.
 
  @param instrument CuInstrument object containing instrument to be simulated.
  @param telescope CuTelescope object containing telescope to be simulated.
  @param source CuSource object containing source definitions.
  @param atmosphere CuAtmosphere object containing atmosphere parameters.
  @param nTimes number of time evaluations in simulation.

  @return BT Array of two dim3 objects, containing number of blocks per grid and number of threads per block.
 */
__host__ void initCUDA(Instrument *instrument, 
        Telescope *telescope, 
        Source *source, 
        Atmosphere *atmosphere, 
        int nTimes,
        int num_stage) 
{
    float dt = 1. / instrument->f_sample;
    float GR_factor = 2 * instrument->delta / instrument->eta_pb;
    float sqrt_samp = sqrtf(0.5 / dt); // Constant term needed for noise calculation
     
    // OBSERVATION-INSTRUMENT PARAMETERS
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cdt), &dt, sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cf_sample), &(instrument->f_sample), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(csqrt_samp), &sqrt_samp, sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cGR_factor), &GR_factor, sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnt), &nTimes, sizeof(int)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnf_ch), &(instrument->nf_ch), sizeof(int)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnum_stage), &num_stage, sizeof(int)) );
    
    // ATMOSPHERE PARAMETERS
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(ch_column), &(atmosphere->h_column), sizeof(float)) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cv_wind), &(atmosphere->v_wind), sizeof(float)) );
}

__global__ void calc_traces_rng(float *az_scan, 
                                float *el_scan, 
                                ArrSpec x_atm,
                                ArrSpec y_atm,
                                float *pwv_screen,
                                float *az_trace,
                                float *el_trace,
                                float *pwv_trace,
                                float *time_trace,
                                hiprandState *state,
                                unsigned long long int seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;                                 
                                                                                     
    if (idx < cnt) 
    {
        if (!seed)
        {
            seed = clock64();
        }

        seed += idx;

        // FLOATS                                                                    
        float time_point;  // Timepoint for thread in simulation.                   
        float pwv_point;   // Container for storing interpolated PWV values.        
        float az_point, el_point;
        float x_point, y_point;
                                                                                     
        time_point = idx * cdt;
        //printf("%.12e\n", time_point);

        az_point = az_scan[idx];
        el_point = el_scan[idx];

        x_point = __tanf(DEG2RAD * az_point) * ch_column + cv_wind * time_point;
        y_point = __tanf(DEG2RAD * el_point) * ch_column;

        interpValue(x_point, y_point,
                    &x_atm, &y_atm, pwv_screen, 0, pwv_point);            
        //printf("%.12e\n", ct_start);
                                                                                     
        hiprand_init(seed, idx, 0, &state[idx]);                                      
        az_trace[idx] = az_point;                                                    
        el_trace[idx] = el_point;
        pwv_trace[idx] = pwv_point;
        time_trace[idx] = time_point + ct_start*cdt;
    }
}

/**
  Main simulation kernel. This is where the magic happens.

  @param eta_cascade Array containing the transmission efficiency of each stage in the cascadei, including the final filterbank stage.
  @param psd_cascade Array containing the parasitic power spectral density of each stage in the cascade, including the final filterbank stage.
  @param num_stages Number of cascade stages, excluding the initial pass of the source signal through the atmosphere and the final filterbank stage.
  @param sigout Array for storing output power, for each channel, for each time, in SI units.
  @param azout Array containing Azimuth coordinates as function of time.
  @param elout Array containing Elevation coordinates as function of time.
  @param flagout Array for storing wether beam is in chop A or B, in nod AB or BA.
  @param PWV_trace Array containing PWV value of atmosphere as seen by telescope over observation, in millimeters.
  @param eta_atm Array with transmission parameters as function of freqs_atm and PWV_atm.
  @param source Array containing source intensity, as function of azsrc, elsrc and freqs_src, in SI units.
 */
__global__ void calc_power(float *az_trace, 
                            float *el_trace, 
                            float *pwv_trace,
                            ArrSpec f_atm, 
                            ArrSpec pwv_atm, 
                            ArrSpec az_src, 
                            ArrSpec el_src,
                            ArrSpec f_src,
                            float *eta_cascade,
                            float *psd_cascade,
                            float *eta_atm,
                            float *sigout, 
                            float *nepout, 
                            float *source,
                            hiprandState *state) 
{
    

    int idx = blockIdx.x * blockDim.x + threadIdx.x; 

    if (idx < cnt) {
        ///////////////////////////////////////
        // DEFINITIONS OF REGISTER VARIABLES //
        ///////////////////////////////////////
        // FLOATS
        float I_nu;             // Specific intensity of source.
        float t, u;             // Interpolation factors
        float eta_atm_interp;   // Interpolated eta_atm, over frequency and PWV
        float freq;             // Bin frequency
        float psd_in;           // Local variable for storing PSD.
        float psd_in_k;         // Local variable for calculating psd per channel
        float eta_kj;           // Filter efficiency for bin j, at channel k.
        float eta_ap;           // Local variable for storing aperture efficiency
        float sigfactor;        // Factor for calculating power. Perform outside of channel loop for speed.
        float csc_el;           // Cosecant of elevation angle.
        float psd_parasitic_use;
        float temp1, temp2, temp3;
        float psd_atm;

        // INTEGERS
        int x0y0, x1y0, x0y1, x1y1; // Indices for interpolation

        hiprandState locstate = state[idx];

        temp1 = az_trace[idx];
        temp2 = el_trace[idx];
        temp3 = pwv_trace[idx];

        csc_el = 1. / __sinf(DEG2RAD * temp2);

        int iAz = floorf((temp1 - az_src.start) / az_src.step);
        int iEl = floorf((temp2 - el_src.start) / el_src.step);

        float az_src_max = az_src.start + az_src.step * (az_src.num - 1);
        float el_src_max = el_src.start + el_src.step * (el_src.num - 1);

        bool offsource = ((temp1 < az_src.start) or (temp1 > az_src_max)) or 
                         ((temp2 < el_src.start) or (temp2 > el_src_max));

        // This can be improved quite alot...
        if(offsource) 
        {
            temp1 = az_src_max;
            temp2 = el_src_max;
        }
        
        x0y0 = f_src.num * (iAz + iEl * az_src.num);
        x1y0 = f_src.num * (iAz + 1 + iEl * az_src.num);
        x0y1 = f_src.num * (iAz + (iEl+1) * az_src.num);
        x1y1 = f_src.num * (iAz + 1 + (iEl+1) * az_src.num);
        
        t = (temp1 - (az_src.start + az_src.step*iAz)) / az_src.step;
        u = (temp2 - (el_src.start + el_src.step*iEl)) / el_src.step;
        
        time_wrt_to(idx, 0);
        // Hier ongeveer starten met loopen over f_src
        for (int idy=0; idy<f_src.num; idy++)
        {
            I_nu = (1-t)*(1-u) * source[x0y0 + idy];
            I_nu += t*(1-u) * source[x1y0 + idy];
            I_nu += (1-t)*u * source[x0y1 + idy];
            I_nu += t*u * source[x1y1 + idy];

            freq = f_src.start + f_src.step * idy;

            interpValue(temp3, freq,
                        &pwv_atm, &f_atm,
                        eta_atm, 0, eta_atm_interp);

            eta_ap = tex1Dfetch(tex_eta_ap, idy); 

            eta_atm_interp = __powf(eta_atm_interp, csc_el);
            psd_atm = tex1Dfetch(tex_psd_atm, idy);

            // Initial pass through atmosphere
            psd_in = eta_ap * I_nu * CL*CL / (freq*freq); 
            psd_in = rad_trans(psd_in, eta_atm_interp, psd_atm);

            // Radiative transfer cascade
            #pragma unroll 
            for (int n=0; n<cnum_stage; n++) 
            {
                psd_parasitic_use = psd_cascade[idy + n*f_src.num];
                if (psd_parasitic_use < 0) 
                {
                    psd_parasitic_use = eta_atm_interp * psd_atm;
                }

                psd_in = rad_trans(psd_in, eta_cascade[idy + n*f_src.num], psd_parasitic_use);
            }

            temp1 = eta_cascade[idy + cnum_stage*f_src.num];
            temp2 = psd_cascade[idy + cnum_stage*f_src.num];

            if(idy==0 and idx== 0){printf("%.12e\n", psd_in);}
            #pragma unroll 

            for(int k=0; k<cnf_ch; k++) {
                if(idy==0 and k==2 and idx== 0){printf("%.12e\n", psd_in);}
                eta_kj = tex1Dfetch( tex_filterbank, k*f_src.num + idy) * temp1;
                psd_in_k = rad_trans(psd_in, eta_kj, temp2);

                sigfactor = psd_in_k * f_src.step; // Note that psd_in already has the eta_kj incorporated!

                sigout[k*cnt + idx] += sigfactor; 
                nepout[k*cnt + idx] += sigfactor * (HP * freq + eta_kj * psd_in_k + cGR_factor); 
            }
        }
        
        time_wrt_to(idx, 0);
        #pragma unroll 
        for(int k=0; k<cnf_ch; k++) {
            temp1 = sqrtf(2 * nepout[k*cnt + idx]) * csqrt_samp;
            temp2 = temp1 * hiprand_normal(&locstate);

            sigout[k*cnt + idx] += temp2;
            state[idx] = locstate;
        }
    }
}

/**
  Run a gateau simulation.
 
  This function is exposed to the ctypes interface and can be called from Python..
 
  @param instrument CuInstrument object containing instrument to be simulated.
  @param telescope CuTelescope object containing telescope to be simulated.
  @param atmosphere CuAtmosphere object containing atmosphere parameters.
  @param source CuSource object containing source definitions.
  @param nTimes Number of time evaluations in simulation.
 */
void run_gateau(Instrument *instrument, 
                     Telescope *telescope, 
                     Atmosphere *atmosphere, 
                     Source *source, 
                     Cascade *cascade,
                     int nTimesTotal, 
                     char *outpath,
                     long long int seed) 
{
    // FLOATS
    float *d_sigout;        // Device pointer for output power array
    float *d_nepout;        // Device pointer for output power array
    float *d_I_nu;          // Device pointer for source intensities
    float *d_az_trace;
    float *d_el_trace;
    float *d_time_trace;
    float *d_pwv_trace;
    
    // INTEGERS
    int nffnt;              // Number of filter frequencies times number of time evaluations
    int nf_src;             // Number of frequency points in source.
    int numSMs;             // Number of streaming multiprocessors on GPU
    int nBlocks1D;          // Number of 1D blocks, in terms of number of SMs

    // OTHER DECLARATIONS
    dim3 blockSize1D;       // Size of 1D block (same as nThreads1D, but dim3 type)
    dim3 gridSize1D;        // Number of 1D blocks per grid

    // ALLOCATE ARRAY SPECIFICATION COPIES
    struct ArrSpec f_src = source->f_spec;
    struct ArrSpec az_src = source->az_src_spec;
    struct ArrSpec el_src = source->el_src_spec;
    
    struct ArrSpec f_atm;
    struct ArrSpec pwv_atm;
    float *eta_atm;

    hiprandState *devstates;

    readEtaATM<float, ArrSpec>(&eta_atm, &pwv_atm, &f_atm);
    
    std::string str_path(atmosphere->path);
    std::string str_outpath(outpath);

    int *meta;
    readAtmMeta(&meta, str_path);

    // Calculate lengths of x and y of single screen
    float lx = meta[1]*atmosphere->dx;              // Length of a single screen along x, in meters
    float ly = meta[2]*atmosphere->dy;              // Length of a single screen along y, in meters
    float lx_av = lx - ly;                          // Available length along x, taking into account center of screen
    float t_obs_av = lx_av / atmosphere->v_wind;    // Max available time per screen

    float timeTotal = nTimesTotal / instrument->f_sample;       // Total time required for simulation

    int nJobs = ceil(timeTotal / t_obs_av);                     // Total number of times kernel needs to be run
    int nTimesScreen = floor(t_obs_av * instrument->f_sample);  // Number of time evaluations available per atmosphere screen. Floored to be safe.

    struct ArrSpec x_atm;
    struct ArrSpec y_atm;

    x_atm.start = -ly/2;
    x_atm.step = atmosphere->dx;
    x_atm.num = meta[1];
    
    y_atm.start = -ly/2;
    y_atm.step = atmosphere->dy;
    y_atm.num = meta[2];

    // Initialize constant memory
    initCUDA(instrument, telescope, source, atmosphere, nTimesScreen, cascade->num_stage); 

    nf_src = f_src.num; // Number of spectral points in source
    
    gpuErrchk( hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0) );

    // TiEMPO2 prefers larger L1 cache over shared memory.
    gpuErrchk( hipDeviceSetCacheConfig(hipFuncCachePreferL1) );
    
    float freq;    // Frequency, used for initialising background sources.

    // Allocate cascade arrays
    std::vector<float> psd_atm(nf_src);

    for(int j=0; j<nf_src; j++)
    {
        freq = f_src.start + f_src.step * j;
        
        psd_atm[j] = get_jn_noise(atmosphere->T_atm, freq); 
    }
    
    float *d_psd_atm;

    gpuErrchk( hipMalloc((void**)&d_psd_atm, nf_src * sizeof(float)) );
    gpuErrchk( hipMemcpy(d_psd_atm, psd_atm.data(), nf_src * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipBindTexture((size_t)0, tex_psd_atm, d_psd_atm, nf_src * sizeof(float)) );
    
    // Allocate cascade arrays
    float *d_eta_cascade, *d_psd_cascade;
    gpuErrchk( hipMalloc((void**)&d_eta_cascade, nf_src * (cascade->num_stage + 1) * sizeof(float)) );
    gpuErrchk( hipMalloc((void**)&d_psd_cascade, nf_src * (cascade->num_stage + 1) * sizeof(float)) );
    gpuErrchk( hipMemcpy(d_eta_cascade, cascade->eta_cascade, nf_src * (cascade->num_stage + 1) * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy(d_psd_cascade, cascade->psd_cascade, nf_src * (cascade->num_stage + 1) * sizeof(float), hipMemcpyHostToDevice) );
    
    // Allocate and copy telescope arrays
    float *d_eta_ap;
    gpuErrchk( hipMalloc((void**)&d_eta_ap, nf_src * sizeof(float)) );
    gpuErrchk( hipMemcpy(d_eta_ap, telescope->eta_ap, nf_src * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipBindTexture((size_t)0, tex_eta_ap, d_eta_ap, nf_src * sizeof(float)) );

    float *d_az_scan, *d_el_scan;
    gpuErrchk( hipMalloc((void**)&d_az_scan, nTimesTotal * sizeof(float)) );
    gpuErrchk( hipMemcpy(d_az_scan, telescope->az_scan, nTimesTotal * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipMalloc((void**)&d_el_scan, nTimesTotal * sizeof(float)) );
    gpuErrchk( hipMemcpy(d_el_scan, telescope->el_scan, nTimesTotal * sizeof(float), hipMemcpyHostToDevice) );

    // Allocate and copy atmosphere arrays
    float *d_eta_atm;
    int neta_atm = f_atm.num * pwv_atm.num;
    
    gpuErrchk( hipMalloc((void**)&d_eta_atm, neta_atm * sizeof(float)) );
    gpuErrchk( hipMemcpy(d_eta_atm, eta_atm, neta_atm * sizeof(float), hipMemcpyHostToDevice) );
    
    delete[] eta_atm;

    // Allocate and copy instrument arrays
    float *d_filterbank;
    int nfilterbank = nf_src * instrument->nf_ch;
    gpuErrchk( hipMalloc((void**)&d_filterbank, nfilterbank * sizeof(float)) );
    gpuErrchk( hipMemcpy(d_filterbank, instrument->filterbank, nfilterbank * sizeof(float), hipMemcpyHostToDevice) );
    gpuErrchk( hipBindTexture((size_t)0, tex_filterbank, d_filterbank, nfilterbank * sizeof(float)) );
    
    gpuErrchk( hipMalloc((void**)&d_I_nu, source->nI_nu * sizeof(float)) );
    gpuErrchk( hipMemcpy(d_I_nu, source->I_nu, source->nI_nu * sizeof(float), hipMemcpyHostToDevice) );

    std::string datp;

    // Loop starts here
    printf("\033[92m");
    int idx_wrap = 0;
    int time_counter = 0;
    float ftime_counter = 0.;
    for(int idx=0; idx<nJobs; idx++) {
        if (idx_wrap == meta[0]) {
            idx_wrap = 0;
        }

        if (idx == (nJobs - 1)) {
            nTimesScreen = nTimesTotal - nTimesScreen*(nJobs-1);
        }
        printf("%.12e\n", ftime_counter);

        gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(ct_start), &ftime_counter, sizeof(float)) );

        time_counter += nTimesScreen;
        ftime_counter = static_cast<float>(time_counter);

        printf("*** Progress: %d / 100 ***\r", time_counter*100 / nTimesTotal);
        fflush(stdout);

        nffnt = instrument->nf_ch * nTimesScreen; // Number of elements in single-screen output.
        gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(cnt), &nTimesScreen, sizeof(int)) );
        
        nBlocks1D = ceilf((float)nTimesScreen / NTHREADS1D / numSMs);
        blockSize1D = NTHREADS1D;
        gridSize1D = nBlocks1D*numSMs;

        // Allocate output arrays
        gpuErrchk( hipMalloc((void**)&d_az_trace, nTimesScreen * sizeof(float)) );
        gpuErrchk( hipMalloc((void**)&d_el_trace, nTimesScreen * sizeof(float)) );
        gpuErrchk( hipMalloc((void**)&d_pwv_trace, nTimesScreen * sizeof(float)) );
        gpuErrchk( hipMalloc((void**)&d_time_trace, nTimesScreen * sizeof(float)) );
        gpuErrchk( hipMalloc((void**)&d_sigout, nffnt * sizeof(float)) );
        gpuErrchk( hipMalloc((void**)&d_nepout, nffnt * sizeof(float)) );
        gpuErrchk( hipMalloc((void**)&devstates, nTimesScreen * sizeof(hiprandState)) );

        // Allocate PWV screen now, delete CUDA allocation after first kernel call
        float *pwv_screen;
        float *d_pwv_screen;
        
        int npwv_screen = x_atm.num * y_atm.num;
        
        //hiprandState *devStates;
        //gpuErrchk( hipMalloc((void **)&devStates, nTimesScreen * sizeof(hiprandState)) );

        datp = std::to_string(idx_wrap) + ".datp";
        readAtmScreen<float, ArrSpec>(&pwv_screen, &x_atm, &y_atm, str_path, datp);
        
        gpuErrchk( hipMalloc((void**)&d_pwv_screen, npwv_screen * sizeof(float)) );
        gpuErrchk( hipMemcpy(d_pwv_screen, pwv_screen, npwv_screen * sizeof(float), hipMemcpyHostToDevice) );
       
        calc_traces_rng<<<gridSize1D, blockSize1D>>>(d_az_scan,
                                                     d_el_scan,
                                                     x_atm,
                                                     y_atm,
                                                     d_pwv_screen,
                                                     d_az_trace,
                                                     d_el_trace,
                                                     d_pwv_trace,
                                                     d_time_trace,
                                                     devstates,
                                                     seed);

        gpuErrchk( hipDeviceSynchronize() );
        gpuErrchk( hipFree(d_pwv_screen) );

        // CALL TO MAIN SIMULATION KERNEL
        calc_power<<<gridSize1D, blockSize1D>>>(d_az_trace,
                                                  d_el_trace,
                                                  d_pwv_trace,
                                                  f_atm, 
                                                  pwv_atm, 
                                                  az_src, 
                                                  el_src,
                                                  f_src,
                                                  d_eta_cascade,
                                                  d_psd_cascade, 
                                                  d_eta_atm,
                                                  d_sigout,
                                                  d_nepout,
                                                  d_I_nu,
                                                  devstates);
        
        gpuErrchk( hipDeviceSynchronize() );
        
        gpuErrchk( hipFree(devstates) );
        gpuErrchk( hipFree(d_nepout) );
        gpuErrchk( hipFree(d_pwv_trace) );
        
        // ALLOCATE STRINGS FOR WRITING OUTPUT
        std::string signame = std::to_string(idx) + "signal.out";
        std::string azname = std::to_string(idx) + "az.out";
        std::string elname = std::to_string(idx) + "el.out";
        std::string timename = std::to_string(idx) + "time.out";

        std::vector<float> sigout(nffnt);
        std::vector<float> azout(nTimesScreen);
        std::vector<float> elout(nTimesScreen);
        std::vector<float> timeout(nTimesScreen);

        gpuErrchk( hipMemcpy(sigout.data(), d_sigout, nffnt * sizeof(float), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(azout.data(), d_az_trace, nTimesScreen * sizeof(float), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(elout.data(), d_el_trace, nTimesScreen * sizeof(float), hipMemcpyDeviceToHost) );
        gpuErrchk( hipMemcpy(timeout.data(), d_time_trace, nTimesScreen * sizeof(float), hipMemcpyDeviceToHost) );

        write1DArray<float>(sigout, str_outpath, signame);
        write1DArray<float>(azout, str_outpath, azname);
        write1DArray<float>(elout, str_outpath, elname);
        write1DArray<float>(timeout, str_outpath, timename);
        
        gpuErrchk( hipFree(d_sigout) );
        gpuErrchk( hipFree(d_az_trace) );
        gpuErrchk( hipFree(d_el_trace) );
        gpuErrchk( hipFree(d_time_trace) );

        idx_wrap++;
    }
    gpuErrchk( hipDeviceReset() );
    printf("\033[0m\n");
}

